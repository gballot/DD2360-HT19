#include "hip/hip_runtime.h"
#include <stdio.h>			// For use of the printf function
#include <sys/time.h>		// For use of gettimeofday function

#define NUM_TIMESTEPS 10000
#define ABS(a) ((a) < 0 ? -(a) : (a))
#define DT 1

int NUM_PARTICLES;	// # of particles to simulate, equivalent to # of threads
int BLOCK_SIZE;		// Threads PER block

// Gravity field
float3 field = (float3) {0.f, 0.f, 9.8f};

// Structure for the particles
typedef struct {
  float3 position;
  float3 velocity;
} Particle;

/**
 * Can use multiple qualifiers to specify where a function will run in order
 * to reuse code that needs to be run on both host and device.
 * Change the position of the given particle based on its velocity using the
 * formula `new_position.coord = old_position.coord + velocity.coord` where
 * coord is x, y and z.
 *
 * @param particle	Particle for which a position update will be performed
 */
__host__ __device__ void updatePosition(Particle *particle) {
  particle->position.x = particle->position.x + particle->velocity.x * DT;
  particle->position.y = particle->position.y + particle->velocity.y * DT;
  particle->position.z = particle->position.z + particle->velocity.z * DT;
}

/**
 * Update the velocity of the given particle according to a field that specifies
 * the rate of change for each dimension of the particle's velocity
 *
 * @param particle	Particle for which a velocity update will be performed
 * @param field		Rate of change for each dimension (x, y, z) of a velocity
 */
__host__ __device__ void updateVelocity(Particle *particle, float3 field) {
  particle->velocity.x = particle->velocity.x + field.x * DT;
  particle->velocity.y = particle->velocity.y + field.y * DT;
  particle->velocity.z = particle->velocity.z + field.z * DT;

}

/**
 * Device implementation for the simulation of moving particles
 *
 * @param particles			List of particles for which to simulate movement
 * @param field				Values specifying the rate of change for a
 *							particle's velocity in each dimension
 * @param num_particles		# of particles, used to determine how many threads
 *							to give work if too many threads are initiated
 */
__global__ void simulateParticlesKernel(Particle *particles, float3 field,
    int num_particles) {

	// Unique ID of the current thread to determine what work to compute
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	// This thread has no work to do, exit
	if (threadId > num_particles) return;

  // Get the right particle
  Particle *particle = particles + threadId;

  // Update velocity first
  updateVelocity(particle, field);

  // Update position
  updatePosition(particle);
}

/**
 * Fill the given array with n random floats.
 *
 * @param array	Array to populate with floats.
 * @param n		Number of floats to populate the array with.
 */
void populateParticleArray(Particle *particles, int n) {
  Particle particle;

	for (int index = 0; index < n; index++) {
		// Generate random particles
    particle.position.x = 10.0 * ((float) rand() / (float) RAND_MAX);
    particle.position.y = 10.0 * ((float) rand() / (float) RAND_MAX);
    particle.position.z = 10.0 * ((float) rand() / (float) RAND_MAX);
    particle.velocity.x = 1.0 * ((float) rand() / (float) RAND_MAX);
    particle.velocity.y = 1.0 * ((float) rand() / (float) RAND_MAX);
    particle.velocity.z = 1.0 * ((float) rand() / (float) RAND_MAX);

		particles[index] = particle;
	}
}

// Entry point into the program, run each implementation of simulation and compare
// the results
int main(int argc, char **argv) {
  char *file_path;
  FILE *out_file = 0;
  bool usePinnedMemory = false;

  if (argc != 1 && argc != 2) {
    printf("Usage: %s <num_particles>\n", argv[0]);
    exit(-1);
  } else {
    NUM_PARTICLES = atoi(argv[1]);
	if (argc == 3) {
		usePinnedMemory = true;
	}
  }

	// Allocate memory on the host
	Particle *hostParticles;
	if (usePinnedMemory) {
		hipHostMalloc(&hostParticles, NUM_PARTICLES * sizeof(Particle));
	} else {
		hostParticles = (Particle *) malloc(NUM_PARTICLES * sizeof(Particle));
	}

	// Allocate memory on the device
	Particle *devParticles;
	hipMalloc(&devParticles, NUM_PARTICLES * sizeof(Particle));

	// Fill hostParticles arrays with random floats
	populateParticleArray(hostParticles, NUM_PARTICLES);

	// After each timestep, copy particle results back to the CPU
	for (int timestep = 0; timestep < NUM_TIMESTEPS; timestep++) {
		// Copy hostParticles onto the GPU
		hipMemcpy(devParticles, hostParticles,
			NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);

		// Round-up to the nearest multiple of BLOCK_SIZE that can hold at least
		// NUM_PARTICLES threads
		simulateParticlesKernel <<<(NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE,
			BLOCK_SIZE>>> (devParticles, field, NUM_PARTICLES, NUM_TIMESTEPS);
		
		// Wait until all the threads on the GPU have finished before continuing
		hipDeviceSynchronize();

		// Copy the result of the simulation on the device back to
		// the host into hostParticles
		hipMemcpy(hostParticles, devParticles,
			NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
	}

	// Free the allocated memory!!!
	if (usePinnedMemory) {
		hipHostFree(hostParticles);
	} else {
		free(hostParticles);
	}

	hipFree(devParticles);

	return 0;
}
